#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>

#include "book.h"
#include "gl_helper.h"

#define MAXTHREADS 512u
#define WIDTH 512
#define HEIGHT 512

int divideCeil(int num, int den){
	return (num+den-1)/den;
}
unsigned int nextPowerOf2(unsigned int n){
  unsigned k=0;
  if(n&&!(n&(n-1))){
	  return n;
  }
  while(n!=0){
    n>>=1;
    k++;
  }
  return 1<<k;
}

__device__
static float G=1000.0f;
__device__
float invsqrt(float x){
	long i;
	float x2, y;
	const float threehalfs = 1.5F;
	x2=x*0.5F;
	y=x;
	i=*(long*)&y;                // evil floating point bit level hacking
	i=0x5f3759df-(i>>1);         // what the fuck?
	y=*(float*)&i;
	y=y*(threehalfs-(x2*y*y));   // 1st iteration
    y=y*(threehalfs-(x2*y*y));   // 2nd iteration, this can be removed
	return y;
}
__device__
float3 operator+(const float3& u, const float3& v) {
    return make_float3(u.x+v.x, u.y+v.y, u.z+v.z);
}
__device__
float3 operator-(const float3& u, const float3& v) {
    return make_float3(u.x-v.x, u.y-v.y, u.z-v.z);
}
__device__
float3 operator*(const float3& u, const float d) {
    return make_float3(u.x*d, u.y*d, u.z*d);
}
__device__
float3 operator/(const float3& u, const float d) {
    return make_float3(u.x/d, u.y/d, u.z/d);
}
__device__
float magnitude2(const float3& v) {
    return v.x*v.x+v.y*v.y+v.z*v.z;
}

__global__
void mapMagnitude2(float3 *d_vec, float* d_mag, const int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		d_mag[i]=magnitude2(d_vec[i]);
	}
}
__global__
void reduceMax(float *d_in, float *d_out, const size_t elements)
{   
    int tid=threadIdx.x;
    int gid=blockIdx.x*blockDim.x+tid;
    extern __shared__ float shared[];
	shared[tid]= gid<elements? d_in[gid]: -FLT_MAX;
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s && gid<elements){
            shared[tid]=max(shared[tid], shared[tid+s]);
        }
        __syncthreads();
    }
    if(tid==0){
        d_out[blockIdx.x]=shared[0];
    }
}
float getMax(float *d_in, int n){
	int grid, block=MAXTHREADS;
	float *h_out=new float();
	do{
		grid=(n+block-1)/block;
		if(grid==1){
			block=nextPowerOf2(n);
		}
		reduceMax<<<grid, block, block*sizeof(float)>>>(d_in, d_in, n);
		n=grid;
	}while(grid>1);
	HANDLE_ERROR(hipMemcpy(h_out, d_in, sizeof(float), hipMemcpyDeviceToHost));
	return *h_out;
}

__global__
void setZero(float3 *d_acc, const int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		d_acc[i]=make_float3(0.0f, 0.0f, 0.0f);
	}
}
__global__
void intialState(float* d_mass, float3 *d_pos, float3 *d_vel, float3 *d_acc, const int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float radius=32+(64.0f*i)/n;
		float angle=16*(6.2832f*i)/n;
		float c=cos(angle);
		float s=sin(angle);
		d_pos[i]=make_float3(radius*c+256, radius*s+256, 0.0f);
		d_vel[i]=make_float3(s, -c, 0.0f)/(invsqrt(G*n/128.0f));
		d_acc[i]=make_float3(0.0f, 0.0f, 0.0f);
		d_mass[i]=1.0f;
	}
}
__global__
void potential(float *mass, float3 *d_pos, float *d_pot, const int n){
	extern __shared__ float s_pot[];
	int tid=threadIdx.x;
	int i=blockIdx.x;
	int j=blockIdx.y*blockDim.x+tid;
	if(j>=n || i==j){
		s_pot[tid]=0.0f;
	}else{
		float3 r=d_pos[i]-d_pos[j];
		s_pot[tid]=-G*mass[i]*mass[j]*invsqrt(magnitude2(r));
	}
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s){
            s_pot[tid]=s_pot[tid]+s_pot[tid+s];
        }
        __syncthreads();
    }
	if(tid==0){
		atomicAdd(&(d_pot[i]), s_pot[0]);
    }
}

__global__
void interact(float *mass, float3 *d_pos, float3 *d_acc, const int n){
	extern __shared__ float3 s_acc[];
	int tid=threadIdx.x;
	int i=blockIdx.x;
	int j=blockIdx.y*blockDim.x+tid;
	if(j>=n || i==j){
		s_acc[tid]=make_float3(0.0f, 0.0f, 0.0f);
	}else{
		float3 r=d_pos[j]-d_pos[i];
		float r2=magnitude2(r)+1;
		s_acc[tid]=r*(invsqrt(r2)*G*mass[j]/r2);
	}
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s){
            s_acc[tid]=s_acc[tid]+s_acc[tid+s];
        }
        __syncthreads();
    }
	if(tid==0){
		atomicAdd(&(d_acc[i].x), s_acc[0].x);
		atomicAdd(&(d_acc[i].y), s_acc[0].y);
		atomicAdd(&(d_acc[i].z), s_acc[0].z);
    }
}
__global__
void move(unsigned char *d_bitmap, float *mass, float3 *d_pos, float3 *d_vel, float3 *d_acc, float dt, const int n) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		d_vel[i]=d_vel[i]+d_acc[i]*dt;
		d_pos[i]=d_pos[i]+d_vel[i]*dt;
		int x=(int)d_pos[i].x;
		int y=(int)d_pos[i].y;
		if(x>=0 && x<WIDTH && y>=0 && y<HEIGHT){
			unsigned int m=255;
			int offset=WIDTH*y+x;
			d_bitmap[4*offset+0]=m;
			d_bitmap[4*offset+1]=m;
			d_bitmap[4*offset+2]=m;
			d_bitmap[4*offset+3]=255;
		}
	}
}

void randset(float* d_in, size_t n, float m, float s){
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateNormal(generator, d_in, n, m, s);
	hiprandDestroyGenerator(generator);
}

struct CPUBitmap {
	unsigned char *pixels;
    int x, y;
	bool exit=false;

    void *dataBlock;
    void (*bitmapExit)(void*);

    CPUBitmap(int width, int height) {
		x=width;
        y=height;
		HANDLE_ERROR(hipHostMalloc((void**)&pixels, 4*width*height));
    }
    ~CPUBitmap() {
        delete[] pixels;
    }

    unsigned char* get_ptr( void ) const   { 
		return pixels; 
	}
    static CPUBitmap** get_bitmap_ptr(void) {
        static CPUBitmap *gBitmap;
        return &gBitmap;
    }
	long image_size( void ) const { 
		return 4*x*y; 
	}

	void display_and_exit(void(*e)(void*)=NULL){
        CPUBitmap** bitmap=get_bitmap_ptr();
        *bitmap=this;
        bitmapExit=e;
        // a bug in the Windows GLUT implementation prevents us from
        // passing zero arguments to glutInit()
        int c=1;
        char* dummy="";
        glutInit(&c, &dummy);
        glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
        glutInitWindowSize(x, y);
        glutCreateWindow("bitmap");
        glutDisplayFunc(Draw);
		//glutWindowStatusFunc();
        glutMainLoop();
    }
    
    // static method used for glut callbacks
    static void Close(void){
		CPUBitmap* bitmap=*(get_bitmap_ptr());
		bitmap->exit=true;
	}
	static void Draw(void){
		CPUBitmap* bitmap=*(get_bitmap_ptr());
		size_t size=bitmap->image_size();

		int n=1024;
		float dt, dvmax=4.0f;
		unsigned char *d_bitmap;
		float *d_mass, *d_aux;
		float3 *d_pos, *d_vel, *d_acc;

		HANDLE_ERROR(hipMalloc((void**)&d_bitmap, size));
		HANDLE_ERROR(hipMalloc((void**)&d_aux, n*sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&d_mass, n*sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&d_pos, n*sizeof(float3)));
		HANDLE_ERROR(hipMalloc((void**)&d_vel, n*sizeof(float3)));
		HANDLE_ERROR(hipMalloc((void**)&d_acc, n*sizeof(float3)));

		int block1D=MAXTHREADS;
		int grid1D=divideCeil(n, block1D);
		int bytes=block1D*sizeof(float3);
		dim3 block2D(MAXTHREADS);
		dim3 grid2D(n, divideCeil(n, MAXTHREADS));
		
		int memory=size+n*(3*sizeof(float3)+2*sizeof(float));
		printf("Currently using %d bytes of device global memory\n", memory);
		
		intialState<<<grid1D, block1D>>>(d_mass, d_pos, d_vel, d_acc, n);
		int i=0;
		do{
			HANDLE_ERROR(hipMemsetAsync(d_bitmap, 0, size));
			setZero<<<grid1D, block1D>>>(d_acc, n);
			interact<<<grid2D, block2D, bytes>>>(d_mass, d_pos, d_acc, n);

			mapMagnitude2<<<grid1D, block1D>>>(d_acc, d_aux, n);
			float amax=sqrt(getMax(d_aux, n));
			dt=dvmax/amax;
			hipDeviceSynchronize();

			move<<<grid1D, block1D>>>(d_bitmap, d_mass, d_pos, d_vel, d_acc, dt, n);
			
			HANDLE_ERROR(hipMemcpy(bitmap->pixels, d_bitmap, size, hipMemcpyDeviceToHost));
			glDrawPixels(bitmap->x, bitmap->y, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
			glFlush();
			i++;
		}while(true);
    }
};

int main( void ) {
    CPUBitmap bitmap(WIDTH, HEIGHT);                              
    bitmap.display_and_exit();
}