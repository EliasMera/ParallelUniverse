#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>

#include "book.h"
#include "gl_helper.h"

#define MAXTHREADS 512u
#define WIDTH 512
#define HEIGHT 512

int divideCeil(int num, int den){
	return (num+den-1)/den;
}
unsigned int nextPowerOf2(unsigned int n){
  unsigned k=0;
  if(n&&!(n&(n-1))){
	  return n;
  }
  while(n!=0){
    n>>=1;
    k++;
  }
  return 1<<k;
}

__device__
static float G=0.1f;
__device__
float invsqrt(float x){
	long i;
	float x2, y;
	const float threehalfs = 1.5F;
	x2=x*0.5F;
	y=x;
	i=*(long*)&y;                // evil floating point bit level hacking
	i=0x5f3759df-(i>>1);         // what the fuck?
	y=*(float*)&i;
	y=y*(threehalfs-(x2*y*y));   // 1st iteration
    y=y*(threehalfs-(x2*y*y));   // 2nd iteration, this can be removed
	return y;
}
__device__
float3 operator+(const float3& u, const float3& v) {
    return make_float3(u.x+v.x, u.y+v.y, u.z+v.z);
}
__device__
float3 operator-(const float3& u, const float3& v) {
    return make_float3(u.x-v.x, u.y-v.y, u.z-v.z);
}
__device__
float3 operator*(const float3& u, const float d) {
    return make_float3(u.x*d, u.y*d, u.z*d);
}
__device__
float3 operator/(const float3& u, const float d) {
    return make_float3(u.x/d, u.y/d, u.z/d);
}
__device__
float magnitude2(const float3& v) {
    return v.x*v.x+v.y*v.y+v.z*v.z;
}

__global__
void mapMagnitude2(float3 *d_vec, float* d_mag, const int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		d_mag[i]=magnitude2(d_vec[i]);
	}
}
__global__
void reduceMax(float *d_in, float *d_out, const size_t elements)
{   
    int tid=threadIdx.x;
    int gid=blockIdx.x*blockDim.x+tid;
    extern __shared__ float shared[];
	shared[tid]= gid<elements? d_in[gid]: -FLT_MAX;
    __syncthreads();
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s && gid<elements){
            shared[tid]=max(shared[tid], shared[tid+s]);
        }
        __syncthreads();
    }
    if(tid==0){
        d_out[blockIdx.x]=shared[0];
    }
}
float getMax(float *d_in, int n){
	int grid, block=MAXTHREADS;
	float *h_out=new float();
	do{
		grid=(n+block-1)/block;
		if(grid==1){
			block=nextPowerOf2(n);
		}
		reduceMax<<<grid, block, block*sizeof(float)>>>(d_in, d_in, n);
		n=grid;
	}while(grid>1);
	HANDLE_ERROR(hipMemcpy(h_out, d_in, sizeof(float), hipMemcpyDeviceToHost));
	return *h_out;
}

__global__
void interact(float *mass, float3 *d_pos, float3 *d_acc, const int n){
	extern __shared__ float3 temp[];
	int tid=threadIdx.x;
	int i=blockIdx.x;
	int j=blockIdx.y*blockDim.x+tid;
	if(j>=n || i==j){
		temp[tid]=make_float3(0.0f, 0.0f, 0.0f);
	}else{
		float3 r=d_pos[j]-d_pos[i];
		float rr=magnitude2(r);
		temp[tid]=r*invsqrt(rr)*(G*mass[j]/rr);
	}
    __syncthreads();
	//Reduction
    for(unsigned int s=blockDim.x/2; s>0; s>>=1){
        if(tid<s){
            temp[tid]=temp[tid]+temp[tid+s];
        }
        __syncthreads();
    }
    if(tid==0){
		atomicAdd(&(d_acc[i].x), temp[0].x);
		atomicAdd(&(d_acc[i].y), temp[0].y);
		atomicAdd(&(d_acc[i].z), temp[0].z);
    }
}
__global__
void move(unsigned char *d_bitmap, float *mass, float3 *d_pos, float3 *d_vel, float3 *d_acc, float dt, const int n) {
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i>=n){
		return;
	}
	d_vel[i]=d_vel[i]+d_acc[i]*dt;
	d_pos[i]=d_pos[i]+d_vel[i]*dt;
	int x=(int)d_pos[i].x;
	int y=(int)d_pos[i].y;
	if(x>=0 && x<WIDTH && y>=0 && y<HEIGHT){
		unsigned int m=255;
		int offset=y*WIDTH+x;
		d_bitmap[4*offset+0]=m;
		d_bitmap[4*offset+1]=m;
		d_bitmap[4*offset+2]=m;
		d_bitmap[4*offset+3]=255;
	}
}

__global__
void setParams(float* d_mass, float3 *d_pos, float3 *d_vel, float3 *d_acc, const int n){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<n){
		float3 r=d_pos[i]-make_float3(256, 256, 256);
		float3 theta=make_float3(r.y, -r.x, 0);
		theta=theta*invsqrt(magnitude2(theta));
		//d_vel[i]=theta*invsqrt(2097152.0f/(G*n*magnitude2(r)));
		d_vel[i]=make_float3(0.0f, 0.0f, 0.0f);
		d_acc[i]=make_float3(0.0f, 0.0f, 0.0f);
		d_mass[i]=1;
	}
}
void randset(float* d_in, size_t n, float m, float s){
	hiprandGenerator_t generator;
	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandGenerateNormal(generator, d_in, n, m, s);
	hiprandDestroyGenerator(generator);
}

struct CPUBitmap {
	unsigned char *pixels;
    int x, y;

    void *dataBlock;
    void (*bitmapExit)(void*);

    CPUBitmap(int width, int height) {
		x=width;
        y=height;
		HANDLE_ERROR(hipHostMalloc((void**)&pixels, 4*width*height));
    }
    ~CPUBitmap() {
        delete[] pixels;
    }

    unsigned char* get_ptr( void ) const   { 
		return pixels; 
	}
    static CPUBitmap** get_bitmap_ptr(void) {
        static CPUBitmap *gBitmap;
        return &gBitmap;
    }
	long image_size( void ) const { 
		return 4*x*y; 
	}

	void display_and_exit(void(*e)(void*)=NULL){
        CPUBitmap** bitmap=get_bitmap_ptr();
        *bitmap=this;
        bitmapExit=e;
        // a bug in the Windows GLUT implementation prevents us from
        // passing zero arguments to glutInit()
        int c=1;
        char* dummy="";
        glutInit(&c, &dummy);
        glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
        glutInitWindowSize(x, y);
        glutCreateWindow("bitmap");
        glutDisplayFunc(Draw);
        glutMainLoop();
    }
    
    // static method used for glut callbacks
    static void Draw(void){
		CPUBitmap* bitmap=*(get_bitmap_ptr());
		size_t size=bitmap->image_size();

		int n=1024;
		float dt, dvmax=4.0f;
		unsigned char *d_bitmap;
		float *d_mass, *d_aux;
		float3 *d_pos, *d_vel, *d_acc;

		HANDLE_ERROR(hipMalloc((void**)&d_bitmap, size));
		HANDLE_ERROR(hipMalloc((void**)&d_aux, n*sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&d_mass, n*sizeof(float)));
		HANDLE_ERROR(hipMalloc((void**)&d_pos, n*sizeof(float3)));
		HANDLE_ERROR(hipMalloc((void**)&d_vel, n*sizeof(float3)));
		HANDLE_ERROR(hipMalloc((void**)&d_acc, n*sizeof(float3)));

		int block1D=MAXTHREADS;
		int grid1D=divideCeil(n, block1D);
		dim3 block2D(MAXTHREADS);
		dim3 grid2D(n, divideCeil(n, MAXTHREADS));
		int bytes=MAXTHREADS*sizeof(float3);
		
		randset((float*)d_pos, 3*n, 256, 32);
		setParams<<<grid1D, block1D>>>(d_mass, d_pos, d_vel, d_acc, n);
		do{
			HANDLE_ERROR(hipMemsetAsync(d_bitmap, 0, size));
			interact<<<grid2D, block2D, bytes>>>(d_mass, d_pos, d_acc, n);

			mapMagnitude2<<<grid1D, block1D>>>(d_acc, d_aux, n);
			dt=dvmax/sqrt(getMax(d_aux, n));
			hipDeviceSynchronize();

			move<<<grid1D, block1D>>>(d_bitmap, d_mass, d_pos, d_vel, d_acc, dt, n);
			HANDLE_ERROR(hipMemcpy(bitmap->pixels, d_bitmap, size, hipMemcpyDeviceToHost));
			glDrawPixels(bitmap->x, bitmap->y, GL_RGBA, GL_UNSIGNED_BYTE, bitmap->pixels);
			glFlush();
		}while(true);
    }
};

int main( void ) {
    CPUBitmap bitmap(WIDTH, HEIGHT);                              
    bitmap.display_and_exit();
}